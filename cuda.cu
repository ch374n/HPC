
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono> 
#define N 100 

using namespace std;
using namespace std::chrono;

__global__ void max_kernel(int *arr, int *n) {
	int tid = threadIdx.x; 

	int start = *n * tid; 
	int end = *n * (tid + 1); 

	for(int i = start + 1; i < end; i++) {
		if(arr[start] < arr[i]) {
			arr[start] = arr[i]; 
		}
	}
}



int main() {
	
	int *dev_arr, *dev_n, *arr, n = 5; 
	int pass = 0; 
	auto start = high_resolution_clock::now();

	hipMalloc(&dev_arr, sizeof(int) * N); 
	hipMalloc(&dev_n, sizeof(int)); 

	hipMemcpy(dev_arr, arr, sizeof(int) * N, hipMemcpyHostToDevice); 
	hipMemcpy(dev_n, &n, sizeof(int), hipMemcpyHostToDevice); 

	int len = N; 

	while(len > n) {

		max_kernel<<<1, len / n>>>(dev_arr, dev_n); 

		hipDeviceSynchronize(); 

		hipMemcpy(arr, dev_arr, sizeof(int) * N, hipMemcpyDeviceToHost); 


		cout<<"Pass : "<<pass++<<endl; 

		for(int i = 0; i < (len / n); i++) {
			arr[i] = arr[i * n];
			cout<<arr[i]<<"\t"; 
		}

		hipMemcpy(dev_arr, arr, sizeof(int) * N, hipMemcpyHostToDevice); 

		len /= n; 
	}

	for(int i = 1; i < len; i++) {
		if(arr[0] < arr[i]) {
			arr[0] = arr[i]; 
		}
	}

	cout<<"Max : "<<arr[0]<<endl; 

	auto end = high_resolution_clock::now();

	cout<<"Time required : "<<(duration_cast<milliseconds>(end - start).count())<<" ms."<<endl; 

	return 0;
}